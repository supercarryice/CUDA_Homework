#include "hip/hip_runtime.h"
#include<cstdio>
#include<iostream>
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include""
#include<cmath>
#include<time.h>
#include <Windows.h>
#define N 106182300
using namespace std;

void add_with_cpu(double A[], int len) {
	double ans = 0;
	clock_t start, end;
	start = clock();
	for (int i = 0; i < len; i++) {
		ans += A[i];
	}
	end = clock();
	cout << "With cpu:    " << "ans:" << ans << "   " << "time:" << end - start << "ms" << endl;

}

__global__ static void add_with_all_atomic(double *A, int len, double *result) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	while (id < len) {
		atomicAdd(result, A[id]);
		id += gridDim.x * blockDim.x;
	}
}

__global__ static void add_with_few_atomic(double *A, int len, double *result) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0.0;
	while (id < len) {
		temp += A[id];
		id += gridDim.x * blockDim.x;
	}
	atomicAdd(result, temp);
}

__global__ static void add_without_atomic(double *A, double *B, int len) {

	extern __shared__ double cache[];
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	double x = 0;

	if (id < len) {
		x = A[id];
	}
	cache[threadIdx.x] = x;
	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if (threadIdx.x < offset)
			cache[threadIdx.x] += cache[threadIdx.x + offset];

		__syncthreads();
	}
	if (threadIdx.x == 0) {
		B[blockIdx.x] == cache[0];
	}
}




int main() {
	double *A = new double[N];
	double result = 0;
	int len;

	double *dev_A;
	double *dev_result;

	hipMalloc((void**)&dev_A, N * sizeof(double));
	hipMalloc((void**)&dev_result, sizeof(double));

	for (int i = 0; i < N; i++) {
		A[i] = (double)(rand() % 101) / 101;
	}
	result = 0;
	len = N;

	hipMemcpy(dev_A, A, N * sizeof(double),
		hipMemcpyHostToDevice);


	hipEvent_t start, stop;
	float elapsedTime;

	// PART1 All atomic
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	add_with_all_atomic << <64, 64 >> > (dev_A, len, dev_result);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);


	hipMemcpy(&result, dev_result, sizeof(double), hipMemcpyDeviceToHost);


	cout << "With all atomic: " << "ans:" << result << "   " << "time:" << elapsedTime << "ms" << endl;

	//PART2 Few Atomic
	double *dev_result1;
	hipMalloc((void**)&dev_result1, sizeof(double));
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	add_with_few_atomic << <64, 64 >> > (dev_A, len, dev_result1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);


	hipMemcpy(&result, dev_result1, sizeof(double), hipMemcpyDeviceToHost);
	cout << "With few atomic: " << "ans:" << result << "   " << "time:" << elapsedTime << "ms" << endl;


	//part3 
	double *dev_result2;
	hipMalloc((void**)&dev_result2, sizeof(double));
	const int block_size = 512;
	const int num_blocks = (len / block_size) + ((len % block_size) ? 1 : 0);
	double *partial_sums = 0;
	hipMalloc((void**)&partial_sums, sizeof(double));

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	add_without_atomic << <num_blocks, block_size, block_size * sizeof(double) >> > (dev_A, partial_sums, len);
	add_without_atomic << <1, num_blocks, num_blocks * sizeof(double) >> > (partial_sums, partial_sums + num_blocks, num_blocks);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(&dev_result2, partial_sums + num_blocks, sizeof(double), hipMemcpyDeviceToHost);

	cout << "Without atomic: " << "ans:" << result << "   " << "time:" << elapsedTime << "ms" << endl;

	add_with_cpu(A, len);
}